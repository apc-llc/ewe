#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <time.h>
#include <cmath>
#include "CudaBernus.h"

#define CUDA_CALL(x) do { hipError_t err = x; if (( err ) != hipSuccess ) { \
	printf ("Error \"%s\" at %s :%d \n" , hipGetErrorString(err), \
		__FILE__ , __LINE__ ) ; exit(-1); \
}} while (0)


#define CUDA_ERR_CHECK(x) \
    do { hipError_t err = x; if (err != hipSuccess) { \
        fprintf (stderr, "Error \"%s\" at %s:%d \n", \
        hipGetErrorString(err), \
        __FILE__, __LINE__); exit(-1); \
    }} while (0);

// using namespace std;
/***********************************************
  the kernel functions 
***********************************************/
  __device__ __const__ int dngates = 5;


  //! Parameter \\( p \\) from Table 4 in Bernus et al. @todo: Make this value celltype-dependent
  __device__ __const__ double p = 1.0;
  
  //! Parameter \\( v_{\\rm shift} \\) from Table 4 in Bernus et al. @todo: Make this value celltype-dependent
  __device__ __const__ double v_shift = 0.0;
  
  //! Intracellular calcium concentration \\( [\\textrm{Ca}^{2+}]_i \\) from Table 1 in Bernus et al.
  __device__ __const__ double ca_i = 0.0004;
  
  //! Extracellular calcium concentration \\( [\\textrm{Ca}^{2+}]_e \\) from Table 1 in Bernus et al.
  __device__ __const__ double ca_e = 2.0;
  
  //! Intracellular sodium concentration \\( [\\textrm{Na}^{+}]_i \\) from Table 1 in Bernus et al
  __device__ __const__ double na_i = 10.0;
  
  //! Extracellular sodium concentration \\( [\\textrm{Na}^{+}]_e \\) from Table 1 in Bernus et al
  __device__ __const__ double na_e = 138.0;
  
  //! Intracellular potassium concentration \\( [\\textrm{K}^{+}]_i \\) from Table 1 in Bernus et al
  __device__ __const__ double k_i  = 140.0;

  //! Extracellular potassium concentration \\( [\\textrm{K}^{+}]_e \\) from Table 1 in Bernus et al
  __device__ __const__ double k_e  = 4.0;
  
  //! Universal gas constant in \\( \\textrm{J}~\\textrm{mol}^{-1}~\\textrm{K}^{-1} \\); note that \\( \\textrm{J} = \\textrm{C}~\\textrm{V} \\).
  //! Because membrane potential in the Bernus model is expressed in milli-Volt, rescale #R to also be in \\( \\textrm{mV} \\).
  __device__ __const__ double R = 8.3144621*1e3;
  
  //! Absolute temperature (in Kelvin, for value in Celsius see Table 1 in Bernus et al.)
  __device__ __const__ double T = 37.0 + 273.15;
  
  //! Faraday constant in \\( \\textrm{C}~\\textrm{mol}^{-1} \\).
  __device__ __const__ double Fa = 9.64853399*1e4;
  
  //! Equilibrium potential \\( E_{\\textrm Na} \\) in millivolt, pp. H2306 in Bernus et al. @todo Would be nicer with constexpr, but use of log in constexpr is not yet supported by clang++
  __device__  double e_na = 70.148612720557793;
  //(R*T/Fa)*log(na_e/na_i);
  
  //! Equilibrium potential \\( E_{\\textrm Ca} \\) in millivolt, pp. H2306 in Bernus et al. @todo Would be nicer with constexpr, but use of log in constexpr is not yet supported by clang+
  __device__  double e_ca = 1.138180432445409e+02;
  //(R*T/(2.0*Fa))*log(ca_e/ca_i) ;

  //! Equilibrium potential \\( E_{to} \\) in millivolt, pp. H2306 in Bernus et al. @todo Would be nicer with constexpr, but use of log in constexpr is not yet supported by clang++
  __device__  double e_to =  -70.792119356993638;
  //(R*T/Fa)*log( (0.043*na_e + k_e)/(0.043*na_i + k_i) );
  
  //! Equilibrium potential \\( E_{\\textrm K} \\) in millivolt, pp. H2306 in Bernus et al. @todo Would be nicer with constexpr, but use of log in constexpr is not yet supported by clang++
  __device__ double e_k = -95.022562085317546;
  //(R*T/Fa)*log(k_e/k_i);
  
  
  /*
   * Implementation of class functions; kept in header for easier inlining.
   * See Bernus et al. 2002 or https://models.cellml.org/e/5/bernus_wilders_zemlin_verschelde_panfilov_2002.cellml/@@cellml_math
   * for the formulas.
   */

  /*
   * (1) Sodium current i_Na (4 Functions)
   */

// m-gate


static __device__ __inline__ double alpha_m(double V)
{
 return 0.32*(V+47.13)/(1.0 - exp(-0.1*(V+47.13)));
}

static __device__ __inline__ double beta_m(double V)
{ 
 return 0.08*exp(-V/11.0); 
}
// v-gate
static __device__ __inline__ double v_inf(double V)
{ 
 return 0.5*(1.0 - (tanh(7.74 + 0.12*V))); 
}

static __device__ __inline__ double tau_v(double V)
{ 
 return 0.25 + 2.24*( 1.0-(tanh(7.74 + 0.12*V)) )/( 1.0 - tanh(0.07*(92.4+V)) ); 
}

 

 /*
  * (2) Calcium current i_Ca (5 functions)
  */

 // d-gate

static __device__ __inline__ double alpha_d(double V)
{ 
 return 14.98*exp(-0.5*pow( (V-22.36)/16.68, 2.0 ))/(16.68*sqrt(2.0*M_PI)); 
}

static __device__ __inline__ double beta_d(double V)
{ 
 return 0.1471 - 5.3*exp(-0.5*pow( (V-6.27)/14.93, 2.0))/(14.93*sqrt(2.0*M_PI)) ; 
}

static __device__ __inline__ double d_inf(double V)
{ 
 return alpha_d(V)/(alpha_d(V)+beta_d(V)); 
}

// f-gate

static __device__ __inline__ double alpha_f(double V)
{ 
 return 6.87e-3/(1.0 + exp( -(6.1546-V)/6.12) ); 
}

static __device__ __inline__ double beta_f(double V)
{ 
 return (0.069*exp(-0.11*(V+9.825))+0.011)/(1.0 + exp(-0.278*(V+9.825))) + 5.75e-4; 
}

// f_Ca-gate
static __device__ __inline__ double f_ca(double V)
{ 
 return 1.0/(1.0 + ca_i/0.0006); 
}

/*
* (3) Transient outward current i_to (7 functions)
*/

// r-gate

static __device__ __inline__ double  alpha_r(double V)
{ 
 return 0.5266*exp(-0.0166*(V-42.2912))/(1.0 + exp(-0.0943*(V-42.2912))); 
}

static __device__ __inline__ double  beta_r(double V)
{
  return (5.186e-5*V+0.5149*exp(-0.1344*(V-5.0027)))/(1.0 + exp(-0.1348*(V-5.186e-5))); 
}

static __device__ __inline__ double  r_inf(double V)
{ 
 return alpha_r(V)/(alpha_r(V)+beta_r(V)); 
}

// to-gate
static __device__ __inline__ double  alpha_to(double V)
{ 
  return (5.612e-5*V+0.0721*exp(-0.173*(V+34.2531)))/(1.0 + exp(-0.173*(V+34.2531))); 
}

static __device__ __inline__ double  beta_to(double V)
{ 
  return (1.215e-4*V + 0.0767*exp(-1.66e-9*(V+34.0235)))/(1.0 + exp(-0.1604*(V+34.0235))); 
}

static __device__ __inline__ double  tau_to(double V)
{ 
  return 1.0/( p*alpha_to(V) + p*beta_to(V)); 
}

static __device__ __inline__ double  to_inf(double V)
{ return alpha_to(V - v_shift)/( alpha_to(V- v_shift) + beta_to(V - v_shift)); }

/*
* (4) Delayed rectifier potassium current i_K (3 functions)
*/

// X-gate
static __device__ __inline__ double  x_inf(double V)
{ 
  return 0.988/(1.0 + exp(-0.861-0.062*V)); 
}

static __device__ __inline__ double  tau_x_a(double V)
{
 return 40.0*(1.0 - tanh(160.0 + 2.0*V)); 
}

static __device__ __inline__ double  tau_x(double V)
{
  return 240.0*exp(-pow( 25.5+V, 2.0)/156.0) + 182.0*(1.0 + tanh(0.154 + 0.0116*V)) + tau_x_a(V); 
}

 
 /*
  * (5) Inward rectifier potassium current i_K1 (3 functions)
  */

 // K1-gate

static __device__ __inline__ double  alpha_k1(double V)
{ 
  return 0.1/(1.0 + exp(0.06*(V-e_k - 200.0))); 
}

static __device__ __inline__ double  beta_k1(double V)
{
  //NOTE: The e_k1 in Bernus et al. is a typo and should be e_k; cf. cellml.org
  return (3.0*exp(2e-4*(V-e_k+100.0)) + exp(0.1*(V-e_k-10.0)))/( 1.0 + exp(-0.5*(V - e_k)) ); 
}

static __device__ __inline__ double  k1_inf(double V)
{ 
  return alpha_k1(V)/(alpha_k1(V) + beta_k1(V)); 
}

  
/*
* (8) Sodium potassium pump (3 functions)
*/
static __device__ __inline__ double  f_nak(double V)
{
  double sigma = 0.1428*( exp(na_e/67.3) - 1.0 );
  return 1.0/(1.0 + 0.1245*exp(-0.0037*V) + 0.0365*sigma*exp(-0.0037*V)); 
}

static __device__ __inline__ double  f_nak_a(double V)
{ 
  return (1.0/(1.0 + pow( 10.0/na_i, 1.5 )))*( k_e/(k_e+1.5) ); 
}

  /*
   * (9) Sodium calcium pump i_NaCa (1 function)
   */
static __device__ __inline__ double f_naca(double V)
{
  double a = 1.0/( (pow(87.5, 3.0) + pow(na_e, 3.0)) * (1.38+ca_e) * (1.0 + 0.1*exp(-0.024*V)) );
  return a*( pow(na_i, 3.0) * ca_e * exp(0.013*V) - pow( na_e, 3.0)*ca_i*exp(-0.024*V) ); 
} 

//! Index of gating variable \\( m \\) in #gates
__device__ __const__ int m_gate  = 0;
  
  //! Index of gating variable \\( v \\) in #gates
__device__ __const__ int v_gate  = 1;
  
  //! Index of gating variable \\( f \\) in #gates
__device__ __const__ int f_gate  = 2;
  
  //! Index of gating variable \\( to \\) in #gates
__device__ __const__ int to_gate = 3;
  
  //! Index of gating variable \\( x \\) in #gates
__device__ __const__ int x_gate  = 4;
  


 //! Constant \\( g_{\rm Na} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_na   = 16.0;

  //! Constant \\( g_{\rm Ca} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_ca   = 0.064;
  
  //! Constant \\( g_{\rm to} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_to   = 0.4;
  
  //! Constant \\( g_{\rm K} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_k    = 0.019;
  
  //! Constant \\( g_{\textrm{K},1} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_k1   = 3.9;
  
  //! Constant \\( g_{\rm Na,b} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_na_b = 0.001;
  
  //! Constant \\( g_{\rm Ca,b} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_ca_b = 0.00085;
  
  //! Constant \\( g_{\rm Na,K} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_nak  = 1.3;
  
  //! Constant \\( g_{\rm Na,Ca} \\) from Table 1 in Bernus et al.
__device__ __const__ double g_naca = 1000.0;

// Sodium current i_Na
static __device__ __inline__ double i_na(double V, double * gates)
{
  return g_na*pow(gates[m_gate], 3.0)*pow(gates[v_gate], 2.0)*(V - e_na);
}  

// Calcium current i_Ca
static __device__ __inline__ double i_ca(double V,double * gates)
{
  return g_ca*(d_inf(V))*gates[f_gate]*(f_ca(V))*(V-e_ca);
}

// Transient outward current i_to
static __device__ __inline__ double i_to(double V,double * gates)
{  
  return g_to*(r_inf(V))*gates[to_gate]*(V-e_to);
}

// Delated rectifier potassium current i_K
static __device__ __inline__ double i_k(double V, double * gates)
{
  return g_k*pow( gates[x_gate], 2.0)*(V-e_k);
}

// Inward rectifier potassium current i_K1
static __device__ __inline__ double i_k1(double V)
{
  return g_k1*(k1_inf(V))*(V-e_k);
}

// Calcium background current
static __device__ __inline__ double i_b_ca(double V)
{
  return g_ca_b*(V-e_ca);
}

// Sodium background current
static __device__ __inline__ double i_b_na(double V)
{
  return g_na_b*(V - e_na);
}

// Sodium potassium pump
static __device__ __inline__ double i_na_k(double V)
{
  return g_nak*(f_nak(V))*(f_nak_a(V));
}

// Sodium calcium pump
static __device__ __inline__ double i_na_ca(double V)
{
  return g_naca*(f_naca(V));
}


/*
 * To faciliate inlining, the ion current functions are implemented here, in the header file.
 * The keyword 'inline' allow e.g. the use of the -Winline flag for gcc to issue a warning if
 * the compiler was unable to actually inline the respective function.
 */
/*

  gates :
    0 -4 are five gate
    5 : V (not anymore)
    6 : Iion (not anymore)
*/


__global__ void rush_larsen_ionforce( double dt, double * vmem, double * oldgates, double * gates, double * Iion, double qpsize) 
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < qpsize){
      // get the voltage for the current thread
      double V = vmem[idx];

      // the ionforcing part here 
      double * tmp = (oldgates + idx * dngates);
      Iion[idx] = i_na(V, tmp )+i_ca(V, tmp)+i_to(V, tmp)+i_k(V, tmp)+i_k1(V)+i_b_ca(V)+i_b_na(V)+i_na_k(V)+i_na_ca(V);

      // copy the oldgates back into gates
      for (int i=0; i< dngates; i++) {
        gates[idx * dngates +  i] = oldgates[idx * dngates + i];
      }

      // the real rush_larsen_step
      double y_inf;
      double tau_y;
      
      // m-gate
      y_inf = alpha_m(V)/( alpha_m(V) + beta_m(V) );
      tau_y = 1.0/( alpha_m(V) + beta_m(V) );
      gates[idx * dngates + m_gate] *= exp(-dt/tau_y);
      gates[idx * dngates + m_gate] += (1.0 - exp(-dt/tau_y))*y_inf;
      
      // // f-gate
      y_inf = alpha_f(V)/( alpha_f(V) + beta_f(V) );
      tau_y = 1.0/( alpha_f(V) + beta_f(V) );
      gates[idx * dngates + f_gate] *= exp(-dt/tau_y);
      gates[idx * dngates + f_gate] += (1.0 - exp(-dt/tau_y))*y_inf;
      
      // // to-gate
      y_inf = alpha_to(V)/( alpha_to(V) + beta_to(V) );
      tau_y = 1.0/( alpha_to(V) + beta_to(V) );
      gates[idx * dngates + to_gate] *= exp(-dt/tau_y);
      gates[idx * dngates + to_gate] += (1.0 - exp(-dt/tau_y))*y_inf;
      
      // // v-gate
      y_inf = v_inf(V);
      tau_y = tau_v(V);
      gates[idx * dngates + v_gate] *= exp(-dt/tau_y);
      gates[idx * dngates + v_gate] += (1.0 - exp(-dt/tau_y))*y_inf;
      
      // // x-gate
      y_inf = x_inf(V);
      tau_y = tau_x(V);
      gates[idx * dngates + x_gate] *= exp(-dt/tau_y);
      gates[idx * dngates + x_gate] += (1.0 - exp(-dt/tau_y))*y_inf;
  }
  
}

/* 
  input : 
     gates is a pointer to pointer
     qpsize is the # of all quadrature points
*/

__global__ void init(double * gates, double qpsize) {
  // Resting potential of Bernus model
  double Vrest = -90.272;
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < qpsize){
    gates[idx * dngates + m_gate]  = alpha_m(Vrest)/(alpha_m(Vrest) + beta_m(Vrest) );
    gates[idx * dngates + v_gate]  = v_inf(Vrest);
    gates[idx * dngates + f_gate]  = alpha_f(Vrest)/( alpha_f(Vrest) + beta_f(Vrest) );
    gates[idx * dngates + to_gate] = alpha_to(Vrest)/( alpha_to(Vrest) + beta_to(Vrest) );
    gates[idx * dngates + x_gate]  = x_inf(Vrest);  
  }
}


/***********************************************
  end of the kernel functions 
***********************************************/

CudaBernus::CudaBernus()
{ 
  
}

void CudaBernus::initialize(double * _gates, int qpsize)
{ 
  int ngates = 5;
  double * d_gates;

  //h_gates = (double *)malloc( (ngates) * qpsize  * sizeof(double));
  CUDA_ERR_CHECK(hipMalloc((void **)&d_gates,  (ngates) *qpsize* sizeof(double)) );

  int nx =  64;
  int ny =  64;
  int nxb = qpsize/nx/ny;
  if ((qpsize - nxb * nx) > 0)
    nxb++;
  dim3 blocks(nxb, 1 ,1);     // cuda 1.x has 1D and 2D grids, cuda 2.x adds 3D grids
  dim3 threadsPerBlock(nx, ny,1 );  // cuda 1.x has 1D, 2D, and 3D blocks 
  init<<< blocks, threadsPerBlock>>>(d_gates, qpsize);
  CUDA_ERR_CHECK(hipGetLastError());
  //CUDA_ERR_CHECK(hipDeviceSynchronize());
  CUDA_ERR_CHECK(hipMemcpy(_gates, d_gates, (ngates) * sizeof(double), hipMemcpyDeviceToHost));

  // copy the value back to _gates
  // for(int i = 0; i < qpsize; i++){
  //   for(int j = 0 ;j < ngates;j++){
  //     (*_gates)[i][j] = h_gates[i * ngates + j] ;
  //   }
  // }
}


void CudaBernus::rush_larsen_ionforcing(double _dt, 
          double * _vmem, 
          double * _gates_old,
          double * _gates,
          double * _Iion,
          int qpsize) 
{
  
  // double * h_gates, *h_oldgates, *h_vmem, *h_Iion;
  // h_gates = (double *)malloc( (dngates) * qpsize  * sizeof(double));
  // h_oldgates = (double *)malloc( (dngates) * qpsize  * sizeof(double));
  // h_vmem = (double *)malloc( qpsize  * sizeof(double));
  // h_Iion = (double *)malloc( qpsize  * sizeof(double));
  
  // h_gates = &_gates_old[0];
  // h_oldgates = &_gates_old[0];
  // for(int i = 0; i < qpsize; i++){
  //   h_vmem[i] = (*_vmem)[i];
  //   for(int j = 0 ;j < dngates;j++){
  //     h_gates[i * qpsize + j] = _gates[i][j];
  //     h_oldgates[i * qpsize + j] = _gates_old[i][j];
  //   }
  //}

  /*
    -----------------------------------------------------
  */

  clock_t timer = clock();

  /*******************************************
   allocate memory on gpu and copy data to gpu
  *********************************************/

  double * d_gates , * d_oldgates, * d_vmem, * d_Iion;

  CUDA_ERR_CHECK(hipMalloc((void **)&d_gates,  (dngates) *qpsize* sizeof(double)) );
  CUDA_ERR_CHECK(hipMalloc((void **)&d_oldgates,  (dngates) *qpsize* sizeof(double)) );
  CUDA_ERR_CHECK(hipMalloc((void **)&d_Iion, qpsize* sizeof(double)) );
  CUDA_ERR_CHECK(hipMalloc((void **)&d_vmem, qpsize* sizeof(double)) );

  CUDA_ERR_CHECK(hipMemcpy(d_gates, _gates,dngates * qpsize * sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERR_CHECK(hipMemcpy(d_oldgates, _gates_old, dngates * qpsize * sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERR_CHECK(hipMemcpy(d_vmem, _vmem,  qpsize * sizeof(double), hipMemcpyHostToDevice));
  
  /*******************************************
    set up the kernel
  *********************************************/

  int nx =  64;
  int ny =  64;
  int nxb = qpsize/nx/ny;
  if ((qpsize - nxb * nx) > 0)
    nxb++;
  dim3 blocks(nxb, 1 ,1);     // cuda 1.x has 1D and 2D grids, cuda 2.x adds 3D grids
  dim3 threadsPerBlock(nx, ny,1 );  // cuda 1.x has 1D, 2D, and 3D blocks 

  // copy the two gates array to device
  CUDA_ERR_CHECK(hipMemcpy(d_gates, _gates, (dngates) * qpsize * sizeof(double), hipMemcpyHostToDevice));
  CUDA_ERR_CHECK(hipMemcpy(d_oldgates, _gates_old, (dngates) * qpsize * sizeof(double), hipMemcpyHostToDevice));

  rush_larsen_ionforce<<< blocks, threadsPerBlock>>>( _dt,  d_vmem, d_oldgates, d_gates, d_Iion, qpsize);
  
  // before copy back, need to synchronize

  CUDA_ERR_CHECK(hipDeviceSynchronize());
  CUDA_ERR_CHECK(hipGetLastError());       
  /*******************************************
    copy data back to Host and time the 
  *********************************************/
  CUDA_ERR_CHECK(hipMemcpy(_gates, d_gates, (dngates) * qpsize * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERR_CHECK(hipMemcpy(_gates_old, d_oldgates, (dngates) * qpsize * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERR_CHECK(hipMemcpy(_Iion ,d_Iion, qpsize * sizeof(double), hipMemcpyDeviceToHost));
  CUDA_ERR_CHECK(hipMemcpy(_vmem ,d_vmem, qpsize * sizeof(double), hipMemcpyDeviceToHost));
  
 
  timer = clock() - timer;
  double time_in_sec = ( (double) timer )/CLOCKS_PER_SEC;
  std::cout << "Total runtime:                       " << time_in_sec << std::endl;


  /****************************************************
    copy data from Host to the corresponding variables
  *****************************************************/

  // for(int i = 0; i < qpsize; i++){
  //   for(int j = 0 ;j < ngates;j++){
  //       (*_gates)[i][j] = h_gates[i * ngates + j] ;
  //       (*_gates)[i][j] = h_gates[i * ngates + j] ;
  //   }
  //   (*_vmem)[i] = h_vmem[i];
  //   (*_Iion)[i] = h_Iion[i];
  // }

  /****************************************************
        Free points both on Host and Deivce
  *****************************************************/

  hipFree(d_gates);
  hipFree(d_oldgates);
  hipFree(d_vmem);
  hipFree(d_Iion);
  // free(h_gates);
  // free(h_oldgates);
  // free(h_vmem);
  // free(h_Iion);


}


